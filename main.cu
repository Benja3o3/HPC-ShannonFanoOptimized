#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <map>
#include <vector>
#include <algorithm>

using namespace std;

__global__ void kernel() {
    // Código del kernel
}

// __global__ void symbolsFrequencyKernel(char *symbols, int *frecuency, int totalSymbols) {
//     int index = threadIdx.x + blockIdx.x * blockDim.x;
//     if (index < totalSymbols) {
//         atomicAdd(&frecuency[symbols[index]], 1);
//     }
// }

int getTotalSymbols(string path);
map<char,int> symbolsFrequency(string path);
vector<pair<char, int>> sortSymbolsByFrecuency(map<char,int> &frecuency);
map<char, vector<bool>> bitsAssignment(vector<pair<char,int>> &vec);
bool sortByValue(const pair<char, int>& a, const pair<char, int>& b);
void shannonFano(vector<pair<char,int>> &vec, int l, int r, map<char, vector<bool>> &code);
int partition(vector<pair<char,int>> &v, int l, int r, map<char, vector<bool>> &code);

int main(int argc, char **argv) {
    if(argc != 2){
		cout << "Error. Debe ejecutarse como ./main 'data/..' " << endl;
		exit(EXIT_FAILURE);
	}
    string filePath = argv[1];
    //Contar simbolos del texto
    int totalSymbols = getTotalSymbols(filePath);
    cout << "Total de simbolos: " << totalSymbols << endl;

    //Calcular la frecuencia de cada simbolo
    map<char,int> frecuency = symbolsFrequency(filePath);

    //Ordenar por frecuencia (de mayor a menor)
    vector<pair<char, int>> vec = sortSymbolsByFrecuency(frecuency);

    //Asignacion de bits a cada simbolo
    map<char, vector<bool>> code = bitsAssignment(vec);

    // // Definir el número de bloques y el número de hilos por bloque
    // int numBlocks = 1;
    // int threadsPerBlock = 1;

    // // Lanzar el kernel en el dispositivo CUDA
    // kernel<<<numBlocks, threadsPerBlock>>>();

    // // Sincronizar el dispositivo CUDA
    // cudaDeviceSynchronize();

    // // Imprimir mensaje de finalización
    // printf("¡Programa en CUDA ejecutado con éxito!\n");

    return 0;
}

bool sortByValue(const pair<char, int>& a, const pair<char, int>& b) {
    return a.second > b.second; // Ordena por los valores de las cadenas
}

int getTotalSymbols(string path) {
    ifstream file(path);
    if (!file) {
        cout << "No se pudo abrir el archivo." << endl;
        return 1;
    }

    file.seekg(0, ios::end);
    streampos tamano = file.tellg();
    file.seekg(0, ios::beg);

    file.close();
    return tamano;
}

map<char,int> symbolsFrequency(string path){
    map<char,int> frecuency;
    time_t start = clock();
    ifstream file(path);
    if (!file) {
        cout << "No se pudo abrir el archivo." << endl;
        return frecuency;
    }

    char symbol;
    while (file.get(symbol)) {
        if (frecuency.find(symbol) == frecuency.end()) {
            frecuency[symbol] = 1;
        } else {
            frecuency[symbol]++;
        }
    }

    file.close();
    time_t end = clock();
    cout << "Tiempo de calculo de la frecuencia de cada simbolos: " << (double)(end - start) / CLOCKS_PER_SEC << " segundos." << endl;
    return frecuency;
}

void shannonFano(vector<pair<char,int>> &vec, int l, int r, map<char, vector<bool>> &code){
    int p;
    if(l<r){
        p = partition(vec, l, r, code);
        shannonFano(vec, l, p-1, code);
        shannonFano(vec, p, r, code);
    }
}

int partition(vector<pair<char,int>> &v, int l, int r, map<char, vector<bool>> &code){
    // cout << "partition" << endl;
	int maxFrecuency = 0;
	//Se calcula la probabilidad maxima de la particion
	for (int x = l; x <= r ; x++){
		maxFrecuency += v[x].second;
	}
	int i = l; //left pos
	int j = r; //right pos
	int izq = v[i].second;
	int der = v[j].second;
	//Se va sumando de hacia el centro para que quede una particion equilibrada
	while(((izq + der) < maxFrecuency)){
		if (der <= izq){
			j--;
			der += v[j].second;
		}
		else{
			i++;
			izq += v[i].second;

		}
	}

	//Asigna "1"(true) a los elementos del lado izquierdo
	// y "0"(false) a los de la derecha
	for(int x = l; x < j; x++){
        if (code.find(v[x].first) == code.end()) {
            code[v[x].first] = vector<bool>(1, true);
        } else {
            code[v[x].first].push_back(true);
        }
	}
	for(int x = j; x <= r; x++){
		if (code.find(v[x].first) == code.end()) {
            code[v[x].first] = vector<bool>(1, false);
        } else {
            code[v[x].first].push_back(false);
        }
	}
	return j;
}

map<char, vector<bool>> bitsAssignment(vector<pair<char,int>> &vec){
    time_t start = clock();
    map<char, vector<bool>> code;
    shannonFano(vec, 0, int(vec.size())-1, code);
    time_t end = clock();
    cout << "Tiempo de asignacion de bits a cada simbolo: " << (double)(end - start) / CLOCKS_PER_SEC << " segundos." << endl;
    return code;
}

vector<pair<char, int>> sortSymbolsByFrecuency(map<char,int> &frecuency){
    time_t start = clock();
    vector<pair<char, int>> vec(frecuency.begin(), frecuency.end());
    sort(vec.begin(), vec.end(), sortByValue);
    time_t end = clock();
    cout << "Tiempo de ordenamiento de los simbolos: " << (double)(end - start) / CLOCKS_PER_SEC << " segundos." << endl;
    return vec;
}
